// RUN: %run_test hipify "%s" "%t" %hipify_args %clang_args

/*
Copyright (c) 2015-present Advanced Micro Devices, Inc. All rights reserved.

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in
all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
THE SOFTWARE.
*/

#include <stdio.h>
// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__
void fn(float* px, float* py) {
  bool a[42];
  __shared__ double b[69];
  for (auto&& x : b) x = *py++;
  for (auto&& x : a) x = *px++ > 0.0;
  for (auto&& x : a) if (x)* --py = *--px;
}

int main() {
  // CHECK: hipFuncCache_t cacheConfig;
  hipFuncCache_t cacheConfig;
  void* func;
  // CHECK: hipFuncSetCacheConfig(reinterpret_cast<const void*>(func), cacheConfig);
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(func), cacheConfig);
  // CHECK: hipFuncAttributes attr{};
  hipFuncAttributes attr{};
  // CHECK: auto r = hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(&fn));
  auto r = hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(&fn));
  // CHECK: if (r != hipSuccess || attr.maxThreadsPerBlock == 0) {
  if (r != hipSuccess || attr.maxThreadsPerBlock == 0) {
    return 1;
  }
  return 0;
}
